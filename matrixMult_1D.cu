#include "hip/hip_runtime.h"
//incomplete
#include <iostream>
#include <cstdlib>
#include <chrono>

__global__ void multiplyMatrix_GPU(int *matrix1, int *matrix2, int *matrix3, int size) 
{
    for(int i = 0; i < size; i++) 
    {
        for(int j=0; j < size; j++) 
        {
            for(int k=0; k < size; k++) 
            {
                matrix3[i * size + j] += matrix1[i * size + k] * matrix2[j + k * size];
            }
        }
    }
}


int main()
{
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);


	int size = 1000;



	int *matrix3 = (int *) malloc(size * size * sizeof(int*));

	auto stopwatchStart = std::chrono::high_resolution_clock::now();
	multiplyMatrix_GPU(matrix1, matrix2, matrix3, size);
	auto stopwatchStop = std::chrono::high_resolution_clock::now();

	std::chrono::duration<float, std::milli> duration_ms = stopwatchStop - stopwatchStart;

	std::cout << std::endl << "Duration of matrix multiplication: " << duration_ms.count() << " milliseconds" << std::endl;

	free(matrix1);
	free(matrix2);
	free(matrix3);

	return 0;
}